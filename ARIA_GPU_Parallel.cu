#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCKLEN 16

//FUNCTION Prototype
__device__ void Add_SubstOdd_Layer(uint8_t* input, uint8_t* key);
__device__ void Add_SubstEven_Layer(uint8_t* input, uint8_t* key);
__device__ void DiffLayer(uint8_t* input);
__device__ void Rot_L(uint8_t* input, uint8_t* output, int num);
__device__ void ROT_XOR(uint8_t* input, uint8_t* op, uint8_t* output, int num);
__global__ void Enc_KEY_Expansion(uint8_t* mk, uint8_t* rk);
__device__ void Round_odd(uint8_t* input, uint8_t* rk);
__device__ void Round_even(uint8_t* input, uint8_t* rk);
__device__ void Final_Round(uint8_t* input, uint8_t* key12, uint8_t* key13);
__device__ void ARIA_EnCrypt(uint8_t* pt, uint8_t* ct, uint8_t* rk);
__device__ void CTR_INC(uint8_t* ICTR);
__global__ void GPU_ARIA_CTR(uint8_t* pt, uint8_t* ct, uint8_t* iv, uint32_t enc_block, uint32_t ptlen, uint8_t* rk);
void printstate(uint8_t* data, int dataLen);


//s-box type1
__constant__ static const uint8_t ARIA_S1[256] = {
	0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
	0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
	0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
	0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
	0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
	0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
	0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
	0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
	0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
	0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
	0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
	0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
	0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
	0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
	0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
	0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};

//s-box type2
__constant__ static const uint8_t ARIA_S2[256] = {
	0xe2, 0x4e, 0x54, 0xfc, 0x94, 0xc2, 0x4a, 0xcc, 0x62, 0x0d, 0x6a, 0x46, 0x3c, 0x4d, 0x8b, 0xd1,
	0x5e, 0xfa, 0x64, 0xcb, 0xb4, 0x97, 0xbe, 0x2b, 0xbc, 0x77, 0x2e, 0x03, 0xd3, 0x19, 0x59, 0xc1,
	0x1d, 0x06, 0x41, 0x6b, 0x55, 0xf0, 0x99, 0x69, 0xea, 0x9c, 0x18, 0xae, 0x63, 0xdf, 0xe7, 0xbb,
	0x00, 0x73, 0x66, 0xfb, 0x96, 0x4c, 0x85, 0xe4, 0x3a, 0x09, 0x45, 0xaa, 0x0f, 0xee, 0x10, 0xeb,
	0x2d, 0x7f, 0xf4, 0x29, 0xac, 0xcf, 0xad, 0x91, 0x8d, 0x78, 0xc8, 0x95, 0xf9, 0x2f, 0xce, 0xcd,
	0x08, 0x7a, 0x88, 0x38, 0x5c, 0x83, 0x2a, 0x28, 0x47, 0xdb, 0xb8, 0xc7, 0x93, 0xa4, 0x12, 0x53,
	0xff, 0x87, 0x0e, 0x31, 0x36, 0x21, 0x58, 0x48, 0x01, 0x8e, 0x37, 0x74, 0x32, 0xca, 0xe9, 0xb1,
	0xb7, 0xab, 0x0c, 0xd7, 0xc4, 0x56, 0x42, 0x26, 0x07, 0x98, 0x60, 0xd9, 0xb6, 0xb9, 0x11, 0x40,
	0xec, 0x20, 0x8c, 0xbd, 0xa0, 0xc9, 0x84, 0x04, 0x49, 0x23, 0xf1, 0x4f, 0x50, 0x1f, 0x13, 0xdc,
	0xd8, 0xc0, 0x9e, 0x57, 0xe3, 0xc3, 0x7b, 0x65, 0x3b, 0x02, 0x8f, 0x3e, 0xe8, 0x25, 0x92, 0xe5,
	0x15, 0xdd, 0xfd, 0x17, 0xa9, 0xbf, 0xd4, 0x9a, 0x7e, 0xc5, 0x39, 0x67, 0xfe, 0x76, 0x9d, 0x43,
	0xa7, 0xe1, 0xd0, 0xf5, 0x68, 0xf2, 0x1b, 0x34, 0x70, 0x05, 0xa3, 0x8a, 0xd5, 0x79, 0x86, 0xa8,
	0x30, 0xc6, 0x51, 0x4b, 0x1e, 0xa6, 0x27, 0xf6, 0x35, 0xd2, 0x6e, 0x24, 0x16, 0x82, 0x5f, 0xda,
	0xe6, 0x75, 0xa2, 0xef, 0x2c, 0xb2, 0x1c, 0x9f, 0x5d, 0x6f, 0x80, 0x0a, 0x72, 0x44, 0x9b, 0x6c,
	0x90, 0x0b, 0x5b, 0x33, 0x7d, 0x5a, 0x52, 0xf3, 0x61, 0xa1, 0xf7, 0xb0, 0xd6, 0x3f, 0x7c, 0x6d,
	0xed, 0x14, 0xe0, 0xa5, 0x3d, 0x22, 0xb3, 0xf8, 0x89, 0xde, 0x71, 0x1a, 0xaf, 0xba, 0xb5, 0x81
};

//inverse of s-box type1
__constant__ static const uint8_t ARIA_RS1[256] = {
	0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
	0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
	0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
	0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
	0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
	0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
	0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
	0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
	0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
	0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
	0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
	0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
	0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
	0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
	0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
	0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d
};

//inverse of s-box type2
__constant__ static const int8_t ARIA_RS2[256] = {
	0x30, 0x68, 0x99, 0x1b, 0x87, 0xb9, 0x21, 0x78, 0x50, 0x39, 0xdb, 0xe1, 0x72, 0x09, 0x62, 0x3c,
	0x3e, 0x7e, 0x5e, 0x8e, 0xf1, 0xa0, 0xcc, 0xa3, 0x2a, 0x1d, 0xfb, 0xb6, 0xd6, 0x20, 0xc4, 0x8d,
	0x81, 0x65, 0xf5, 0x89, 0xcb, 0x9d, 0x77, 0xc6, 0x57, 0x43, 0x56, 0x17, 0xd4, 0x40, 0x1a, 0x4d,
	0xc0, 0x63, 0x6c, 0xe3, 0xb7, 0xc8, 0x64, 0x6a, 0x53, 0xaa, 0x38, 0x98, 0x0c, 0xf4, 0x9b, 0xed,
	0x7f, 0x22, 0x76, 0xaf, 0xdd, 0x3a, 0x0b, 0x58, 0x67, 0x88, 0x06, 0xc3, 0x35, 0x0d, 0x01, 0x8b,
	0x8c, 0xc2, 0xe6, 0x5f, 0x02, 0x24, 0x75, 0x93, 0x66, 0x1e, 0xe5, 0xe2, 0x54, 0xd8, 0x10, 0xce,
	0x7a, 0xe8, 0x08, 0x2c, 0x12, 0x97, 0x32, 0xab, 0xb4, 0x27, 0x0a, 0x23, 0xdf, 0xef, 0xca, 0xd9,
	0xb8, 0xfa, 0xdc, 0x31, 0x6b, 0xd1, 0xad, 0x19, 0x49, 0xbd, 0x51, 0x96, 0xee, 0xe4, 0xa8, 0x41,
	0xda, 0xff, 0xcd, 0x55, 0x86, 0x36, 0xbe, 0x61, 0x52, 0xf8, 0xbb, 0x0e, 0x82, 0x48, 0x69, 0x9a,
	0xe0, 0x47, 0x9e, 0x5c, 0x04, 0x4b, 0x34, 0x15, 0x79, 0x26, 0xa7, 0xde, 0x29, 0xae, 0x92, 0xd7,
	0x84, 0xe9, 0xd2, 0xba, 0x5d, 0xf3, 0xc5, 0xb0, 0xbf, 0xa4, 0x3b, 0x71, 0x44, 0x46, 0x2b, 0xfc,
	0xeb, 0x6f, 0xd5, 0xf6, 0x14, 0xfe, 0x7c, 0x70, 0x5a, 0x7d, 0xfd, 0x2f, 0x18, 0x83, 0x16, 0xa5,
	0x91, 0x1f, 0x05, 0x95, 0x74, 0xa9, 0xc1, 0x5b, 0x4a, 0x85, 0x6d, 0x13, 0x07, 0x4f, 0x4e, 0x45,
	0xb2, 0x0f, 0xc9, 0x1c, 0xa6, 0xbc, 0xec, 0x73, 0x90, 0x7b, 0xcf, 0x59, 0x8f, 0xa1, 0xf9, 0x2d,
	0xf2, 0xb1, 0x00, 0x94, 0x37, 0x9f, 0xd0, 0x2e, 0x9c, 0x6e, 0x28, 0x3f, 0x80, 0xf0, 0x3d, 0xd3,
	0x25, 0x8a, 0xb5, 0xe7, 0x42, 0xb3, 0xc7, 0xea, 0xf7, 0x4c, 0x11, 0x33, 0x03, 0xa2, 0xac, 0x60
};

//AddRoundKey + SubstOdd_Layer(Ȧ�� ���� ġȯ����)
__device__ void Add_SubstOdd_Layer(uint8_t* input, uint8_t* key) {
	*(input + 0) = ARIA_S1[*(input + 0) ^ *(key + 0)];
	*(input + 1) = ARIA_S2[*(input + 1) ^ *(key + 1)];
	*(input + 2) = ARIA_RS1[*(input + 2) ^ *(key + 2)];
	*(input + 3) = ARIA_RS2[*(input + 3) ^ *(key + 3)];
	*(input + 4) = ARIA_S1[*(input + 4) ^ *(key + 4)];
	*(input + 5) = ARIA_S2[*(input + 5) ^ *(key + 5)];
	*(input + 6) = ARIA_RS1[*(input + 6) ^ *(key + 6)];
	*(input + 7) = ARIA_RS2[*(input + 7) ^ *(key + 7)];
	*(input + 8) = ARIA_S1[*(input + 8) ^ *(key + 8)];
	*(input + 9) = ARIA_S2[*(input + 9) ^ *(key + 9)];
	*(input + 10) = ARIA_RS1[*(input + 10) ^ *(key + 10)];
	*(input + 11) = ARIA_RS2[*(input + 11) ^ *(key + 11)];
	*(input + 12) = ARIA_S1[*(input + 12) ^ *(key + 12)];
	*(input + 13) = ARIA_S2[*(input + 13) ^ *(key + 13)];
	*(input + 14) = ARIA_RS1[*(input + 14) ^ *(key + 14)];
	*(input + 15) = ARIA_RS2[*(input + 15) ^ *(key + 15)];
}

//AddRoundKey + SubstOdd_Layer(¦�� ���� ġȯ����)
__device__ void Add_SubstEven_Layer(uint8_t* input, uint8_t* key) {
	*(input + 0) = ARIA_RS1[*(input + 0) ^ *(key + 0)];
	*(input + 1) = ARIA_RS2[*(input + 1) ^ *(key + 1)];
	*(input + 2) = ARIA_S1[*(input + 2) ^ *(key + 2)];
	*(input + 3) = ARIA_S2[*(input + 3) ^ *(key + 3)];
	*(input + 4) = ARIA_RS1[*(input + 4) ^ *(key + 4)];
	*(input + 5) = ARIA_RS2[*(input + 5) ^ *(key + 5)];
	*(input + 6) = ARIA_S1[*(input + 6) ^ *(key + 6)];
	*(input + 7) = ARIA_S2[*(input + 7) ^ *(key + 7)];
	*(input + 8) = ARIA_RS1[*(input + 8) ^ *(key + 8)];
	*(input + 9) = ARIA_RS2[*(input + 9) ^ *(key + 9)];
	*(input + 10) = ARIA_S1[*(input + 10) ^ *(key + 10)];
	*(input + 11) = ARIA_S2[*(input + 11) ^ *(key + 11)];
	*(input + 12) = ARIA_RS1[*(input + 12) ^ *(key + 12)];
	*(input + 13) = ARIA_RS2[*(input + 13) ^ *(key + 13)];
	*(input + 14) = ARIA_S1[*(input + 14) ^ *(key + 14)];
	*(input + 15) = ARIA_S2[*(input + 15) ^ *(key + 15)];
}

//Ȯ�����
__device__ void DiffLayer(uint8_t* input) {
	uint8_t temp[16] = { 0x00, };//��İ��� ��� ��Ƴ��� ����
	*(temp + 0) = *(input + 3) ^ *(input + 4) ^ *(input + 6) ^ *(input + 8) ^ *(input + 9) ^ *(input + 13) ^ *(input + 14);
	*(temp + 1) = *(input + 2) ^ *(input + 5) ^ *(input + 7) ^ *(input + 8) ^ *(input + 9) ^ *(input + 12) ^ *(input + 15);
	*(temp + 2) = *(input + 1) ^ *(input + 4) ^ *(input + 6) ^ *(input + 10) ^ *(input + 11) ^ *(input + 12) ^ *(input + 15);
	*(temp + 3) = *(input + 0) ^ *(input + 5) ^ *(input + 7) ^ *(input + 10) ^ *(input + 11) ^ *(input + 13) ^ *(input + 14);
	*(temp + 4) = *(input + 0) ^ *(input + 2) ^ *(input + 5) ^ *(input + 8) ^ *(input + 11) ^ *(input + 14) ^ *(input + 15);
	*(temp + 5) = *(input + 1) ^ *(input + 3) ^ *(input + 4) ^ *(input + 9) ^ *(input + 10) ^ *(input + 14) ^ *(input + 15);
	*(temp + 6) = *(input + 0) ^ *(input + 2) ^ *(input + 7) ^ *(input + 9) ^ *(input + 10) ^ *(input + 12) ^ *(input + 13);
	*(temp + 7) = *(input + 1) ^ *(input + 3) ^ *(input + 6) ^ *(input + 8) ^ *(input + 11) ^ *(input + 12) ^ *(input + 13);
	*(temp + 8) = *(input + 0) ^ *(input + 1) ^ *(input + 4) ^ *(input + 7) ^ *(input + 10) ^ *(input + 13) ^ *(input + 15);
	*(temp + 9) = *(input + 0) ^ *(input + 1) ^ *(input + 5) ^ *(input + 6) ^ *(input + 11) ^ *(input + 12) ^ *(input + 14);
	*(temp + 10) = *(input + 2) ^ *(input + 3) ^ *(input + 5) ^ *(input + 6) ^ *(input + 8) ^ *(input + 13) ^ *(input + 15);
	*(temp + 11) = *(input + 2) ^ *(input + 3) ^ *(input + 4) ^ *(input + 7) ^ *(input + 9) ^ *(input + 12) ^ *(input + 14);
	*(temp + 12) = *(input + 1) ^ *(input + 2) ^ *(input + 6) ^ *(input + 7) ^ *(input + 9) ^ *(input + 11) ^ *(input + 12);
	*(temp + 13) = *(input + 0) ^ *(input + 3) ^ *(input + 6) ^ *(input + 7) ^ *(input + 8) ^ *(input + 10) ^ *(input + 13);
	*(temp + 14) = *(input + 0) ^ *(input + 3) ^ *(input + 4) ^ *(input + 5) ^ *(input + 9) ^ *(input + 11) ^ *(input + 14);
	*(temp + 15) = *(input + 1) ^ *(input + 2) ^ *(input + 4) ^ *(input + 5) ^ *(input + 8) ^ *(input + 10) ^ *(input + 15);
	//���ó��
	memcpy(input, temp, 16 * sizeof(uint8_t));
}


//128-bit Left ROTATE �Լ�
//ROT_R�� ROT_L(N)=ROT_R(128-N)���� �̿��Ұ�!
__device__ void Rot_L(uint8_t* input, uint8_t* output, int num) {
	uint8_t copy_input[16] = { 0x00, };
	memcpy(copy_input, input, 16 * sizeof(uint8_t));
	int r = num & 0x07;//%8
	int q = num >> 3;
	uint8_t temp[16] = { 0x00, };
	memcpy(temp, input, 16 * sizeof(uint8_t));
	if (r == 0) {
		*(copy_input + 0) = *(temp + (0 + q) % BLOCKLEN);
		*(copy_input + 1) = *(temp + (1 + q) % BLOCKLEN);
		*(copy_input + 2) = *(temp + (2 + q) % BLOCKLEN);
		*(copy_input + 3) = *(temp + (3 + q) % BLOCKLEN);
		*(copy_input + 4) = *(temp + (4 + q) % BLOCKLEN);
		*(copy_input + 5) = *(temp + (5 + q) % BLOCKLEN);
		*(copy_input + 6) = *(temp + (6 + q) % BLOCKLEN);
		*(copy_input + 7) = *(temp + (7 + q) % BLOCKLEN);
		*(copy_input + 8) = *(temp + (8 + q) % BLOCKLEN);
		*(copy_input + 9) = *(temp + (9 + q) % BLOCKLEN);
		*(copy_input + 10) = *(temp + (10 + q) % BLOCKLEN);
		*(copy_input + 11) = *(temp + (11 + q) % BLOCKLEN);
		*(copy_input + 12) = *(temp + (12 + q) % BLOCKLEN);
		*(copy_input + 13) = *(temp + (13 + q) % BLOCKLEN);
		*(copy_input + 14) = *(temp + (14 + q) % BLOCKLEN);
		*(copy_input + 15) = *(temp + (15 + q) % BLOCKLEN);
	}
	else {
		*(copy_input + 0) = *(temp + (0 + q) % BLOCKLEN) << r | (*(temp + (0 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 1) = *(temp + (1 + q) % BLOCKLEN) << r | (*(temp + (1 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 2) = *(temp + (2 + q) % BLOCKLEN) << r | (*(temp + (2 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 3) = *(temp + (3 + q) % BLOCKLEN) << r | (*(temp + (3 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 4) = *(temp + (4 + q) % BLOCKLEN) << r | (*(temp + (4 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 5) = *(temp + (5 + q) % BLOCKLEN) << r | (*(temp + (5 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 6) = *(temp + (6 + q) % BLOCKLEN) << r | (*(temp + (6 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 7) = *(temp + (7 + q) % BLOCKLEN) << r | (*(temp + (7 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 8) = *(temp + (8 + q) % BLOCKLEN) << r | (*(temp + (8 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 9) = *(temp + (9 + q) % BLOCKLEN) << r | (*(temp + (9 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 10) = *(temp + (10 + q) % BLOCKLEN) << r | (*(temp + (10 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 11) = *(temp + (11 + q) % BLOCKLEN) << r | (*(temp + (11 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 12) = *(temp + (12 + q) % BLOCKLEN) << r | (*(temp + (12 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 13) = *(temp + (13 + q) % BLOCKLEN) << r | (*(temp + (13 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 14) = *(temp + (14 + q) % BLOCKLEN) << r | (*(temp + (14 + q + 1) % BLOCKLEN) >> (8 - r));
		*(copy_input + 15) = *(temp + (15 + q) % BLOCKLEN) << r | (*(temp + (15 + q + 1) % BLOCKLEN) >> (8 - r));
	}
	memcpy(output, copy_input, BLOCKLEN * sizeof(uint8_t));
}

//rotate��Ű�� xor���� ����
//parameter ����(parameter �������)
//rotate��ų��, ���ǰ��� xor���ٰ�, �����, rotateȽ��(R_ROTATE��� 128-num)
__device__ void ROT_XOR(uint8_t* input, uint8_t* op, uint8_t* output, int num) {
	uint8_t temp[16] = { 0x00, };
	Rot_L(input, temp, num);
	*(output + 0) = *(temp + 0) ^ *(op + 0);
	*(output + 1) = *(temp + 1) ^ *(op + 1);
	*(output + 2) = *(temp + 2) ^ *(op + 2);
	*(output + 3) = *(temp + 3) ^ *(op + 3);
	*(output + 4) = *(temp + 4) ^ *(op + 4);
	*(output + 5) = *(temp + 5) ^ *(op + 5);
	*(output + 6) = *(temp + 6) ^ *(op + 6);
	*(output + 7) = *(temp + 7) ^ *(op + 7);
	*(output + 8) = *(temp + 8) ^ *(op + 8);
	*(output + 9) = *(temp + 9) ^ *(op + 9);
	*(output + 10) = *(temp + 10) ^ *(op + 10);
	*(output + 11) = *(temp + 11) ^ *(op + 11);
	*(output + 12) = *(temp + 12) ^ *(op + 12);
	*(output + 13) = *(temp + 13) ^ *(op + 13);
	*(output + 14) = *(temp + 14) ^ *(op + 14);
	*(output + 15) = *(temp + 15) ^ *(op + 15);
}


//��ȣȭ ���� Ű������
//mk�� ������Ű -> ������Ű�κ��� 12R rk����
__global__ void Enc_KEY_Expansion(uint8_t* mk, uint8_t* rk) { //rk[13][16] -> rk[208]
	//initial part 
	uint8_t KL[16] = { 0x00, };
	memcpy(KL, mk, 16 * sizeof(uint8_t));

	uint8_t CK1[16] = { 0x51,0x7c,0xc1,0xb7,0x27,0x22,0x0a,0x94,0xfe,0x13,0xab,0xe8,0xfa,0x9a,0x6e,0xe0 };
	uint8_t CK2[16] = { 0x6d,0xb1,0x4a,0xcc,0x9e,0x21,0xc8,0x20,0xff,0x28,0xb1,0xd5,0xef,0x5d,0xe2,0xb0 };
	uint8_t CK3[16] = { 0xdb,0x92,0x37,0x1d,0x21,0x26,0xe9,0x70,0x03,0x24,0x97,0x75,0x04,0xe8,0xc9,0x0e };

	//W0
	uint8_t W0[16] = { 0x00, };
	memcpy(W0, KL, 16 * sizeof(uint8_t));
	//W1
	uint8_t copy_W0[16] = { 0x00, };
	memcpy(copy_W0, W0, 16 * sizeof(uint8_t));
	uint8_t W1[16] = { 0x00, };
	Round_odd(copy_W0, CK1);
	memcpy(W1, copy_W0, 16 * sizeof(uint8_t));
	//W2
	uint8_t W2[16] = { 0x00, };
	uint8_t copy_W1[16] = { 0x00, };
	memcpy(copy_W1, W1, 16 * sizeof(uint8_t));
	Round_even(copy_W1, CK2);
	*(W2 + 0) = *(copy_W1 + 0) ^ *(W0 + 0);
	*(W2 + 1) = *(copy_W1 + 1) ^ *(W0 + 1);
	*(W2 + 2) = *(copy_W1 + 2) ^ *(W0 + 2);
	*(W2 + 3) = *(copy_W1 + 3) ^ *(W0 + 3);
	*(W2 + 4) = *(copy_W1 + 4) ^ *(W0 + 4);
	*(W2 + 5) = *(copy_W1 + 5) ^ *(W0 + 5);
	*(W2 + 6) = *(copy_W1 + 6) ^ *(W0 + 6);
	*(W2 + 7) = *(copy_W1 + 7) ^ *(W0 + 7);
	*(W2 + 8) = *(copy_W1 + 8) ^ *(W0 + 8);
	*(W2 + 9) = *(copy_W1 + 9) ^ *(W0 + 9);
	*(W2 + 10) = *(copy_W1 + 10) ^ *(W0 + 10);
	*(W2 + 11) = *(copy_W1 + 11) ^ *(W0 + 11);
	*(W2 + 12) = *(copy_W1 + 12) ^ *(W0 + 12);
	*(W2 + 13) = *(copy_W1 + 13) ^ *(W0 + 13);
	*(W2 + 14) = *(copy_W1 + 14) ^ *(W0 + 14);
	*(W2 + 15) = *(copy_W1 + 15) ^ *(W0 + 15);
	//W3
	uint8_t W3[16] = { 0x00, };
	uint8_t copy_W2[16] = { 0x00, };
	memcpy(copy_W2, W2, 16 * sizeof(uint8_t));
	Round_odd(copy_W2, CK3);
	*(W3 + 0) = *(copy_W2 + 0) ^ *(W1 + 0);
	*(W3 + 1) = *(copy_W2 + 1) ^ *(W1 + 1);
	*(W3 + 2) = *(copy_W2 + 2) ^ *(W1 + 2);
	*(W3 + 3) = *(copy_W2 + 3) ^ *(W1 + 3);
	*(W3 + 4) = *(copy_W2 + 4) ^ *(W1 + 4);
	*(W3 + 5) = *(copy_W2 + 5) ^ *(W1 + 5);
	*(W3 + 6) = *(copy_W2 + 6) ^ *(W1 + 6);
	*(W3 + 7) = *(copy_W2 + 7) ^ *(W1 + 7);
	*(W3 + 8) = *(copy_W2 + 8) ^ *(W1 + 8);
	*(W3 + 9) = *(copy_W2 + 9) ^ *(W1 + 9);
	*(W3 + 10) = *(copy_W2 + 10) ^ *(W1 + 10);
	*(W3 + 11) = *(copy_W2 + 11) ^ *(W1 + 11);
	*(W3 + 12) = *(copy_W2 + 12) ^ *(W1 + 12);
	*(W3 + 13) = *(copy_W2 + 13) ^ *(W1 + 13);
	*(W3 + 14) = *(copy_W2 + 14) ^ *(W1 + 14);
	*(W3 + 15) = *(copy_W2 + 15) ^ *(W1 + 15);
	//result part
	ROT_XOR(W1, W0, rk, 128 - 19);
	ROT_XOR(W2, W1, rk + 16, 128 - 19);
	ROT_XOR(W3, W2, rk + 32, 128 - 19);
	ROT_XOR(W0, W3, rk + 48, 128 - 19);
	ROT_XOR(W1, W0, rk + 64, 128 - 31);
	ROT_XOR(W2, W1, rk + 80, 128 - 31);
	ROT_XOR(W3, W2, rk + 96, 128 - 31);
	ROT_XOR(W0, W3, rk + 112, 128 - 31);
	ROT_XOR(W1, W0, rk + 128, 61);
	ROT_XOR(W2, W1, rk + 144, 61);
	ROT_XOR(W3, W2, rk + 160, 61);
	ROT_XOR(W0, W3, rk + 176, 61);
	ROT_XOR(W1, W0, rk + 192, 31);
}

//Ȧ�� �����Լ�
__device__ void Round_odd(uint8_t* input, uint8_t* rk) {
	Add_SubstOdd_Layer(input, rk);
	DiffLayer(input);
}

//¦�� �����Լ�
__device__ void Round_even(uint8_t* input, uint8_t* rk) {
	Add_SubstEven_Layer(input, rk);
	DiffLayer(input);
}

//������ ���� �Լ�
__device__ void Final_Round(uint8_t* input, uint8_t* key12, uint8_t* key13) {
	Add_SubstEven_Layer(input, key12);
	//������ ����Ű�� XOR
	*(input + 0) ^= *(key13 + 0);
	*(input + 1) ^= *(key13 + 1);
	*(input + 2) ^= *(key13 + 2);
	*(input + 3) ^= *(key13 + 3);
	*(input + 4) ^= *(key13 + 4);
	*(input + 5) ^= *(key13 + 5);
	*(input + 6) ^= *(key13 + 6);
	*(input + 7) ^= *(key13 + 7);
	*(input + 8) ^= *(key13 + 8);
	*(input + 9) ^= *(key13 + 9);
	*(input + 10) ^= *(key13 + 10);
	*(input + 11) ^= *(key13 + 11);
	*(input + 12) ^= *(key13 + 12);
	*(input + 13) ^= *(key13 + 13);
	*(input + 14) ^= *(key13 + 14);
	*(input + 15) ^= *(key13 + 15);
}


//ARIA128 ��ȣȭ �Լ�
__device__ void ARIA_EnCrypt(uint8_t* pt, uint8_t* ct, uint8_t* rk) {
	uint8_t copy_pt[16] = { 0x00, };
	memcpy(copy_pt, pt, 16 * sizeof(uint8_t));

	//1R~11R
	Round_odd(copy_pt, rk);
	Round_even(copy_pt, rk + 16);
	Round_odd(copy_pt, rk + 32);
	Round_even(copy_pt, rk + 48);
	Round_odd(copy_pt, rk + 64);
	Round_even(copy_pt, rk + 80);
	Round_odd(copy_pt, rk + 96);
	Round_even(copy_pt, rk + 112);
	Round_odd(copy_pt, rk + 128);
	Round_even(copy_pt, rk + 144);
	Round_odd(copy_pt, rk + 160);

	//12R
	Final_Round(copy_pt, rk + 176, rk + 192);

	//���ó��
	memcpy(ct, copy_pt, 16 * sizeof(uint8_t));
}

//CTR�� +1�� ���ִ� �Լ�
__device__ void CTR_INC(uint8_t* ICTR) {
	int cnt_i;
	for (cnt_i = 15; cnt_i > 7; cnt_i--) {
		if (*(ICTR + cnt_i) != 0xff) {
			*(ICTR + cnt_i) += 1;
			break;
		}
		else {
			*(ICTR + cnt_i) = 0x00;
		}
	}
}

//GPU���� ��Ƽ������� ��ȣ������ ���������� ó�����ִ� �ھ� �Լ�
__global__ void GPU_ARIA_CTR(uint8_t* pt, uint8_t* ct, uint8_t* iv, uint32_t enc_block, uint32_t ptlen, uint8_t* rk) {
	int i;
	//CTR IV
	uint8_t ICTR[16] = { 0x00, };
	memcpy(ICTR, iv, 16 * sizeof(uint8_t));

	//temp_pt -> �� �����尡 pt�������� �ڱⰡ ��ȣȭ ����� �ϴ� �Ϻκ��� ���ͼ� ������ ����
	//temp_ctr -> �� �����尡 �ڱⰡ ��ȣȭ ����� �ϴ� CTR���� ������ ����
	uint8_t temp_pt[16] = { 0x00, };
	uint8_t temp_ctr[16] = { 0x00, };

	//Round Key�� global memory -> shared memory 
	//�޸� ���� �ӵ��� ������
	//bank conflict�� �ذ�X
	__shared__ uint8_t roundkey[208];
	memcpy(roundkey, rk, 208);

	//��ȣ���� ����� �� �����忡 �´� ct�� �־��ֱ� ���� index
	int index = (threadIdx.x * 16) + (blockDim.x * blockIdx.x * 16);

	//��� block�� ��������� ���������� ��ȣ�� �Űܼ� �ο�
	int thread_num = threadIdx.x + (blockDim.x * blockIdx.x);

	
	//�� ������ �ڱ� ��ü���� �������ȣ�� �°� CTR_INC ����
	for (i = 0; i < thread_num; i++) {
		CTR_INC(ICTR);
	}

	//�� �����尡 �ڱ����� �˸��� CounTer���� ��ȣȭ
	ARIA_EnCrypt(ICTR, temp_ctr, roundkey);

	//�� ������ pt���� �ڱⰡ ��ȣȭ �ؾ��ϴ� ���� ������ �۾� 
	for (i = 0; i < 16; i++) {
		*(temp_pt + i) = *(pt + (enc_block * i + thread_num));
	}

	//�� ������ �ڱⰡ ��ȣȭ �ؾ��ϴ� ������ xor���ִ� �۾� 
	for (i = 0; i < 16; i++) {
		*(ct + (index + i)) = *(temp_pt + i) ^ *(temp_ctr + i);
	}
}

//GPU������ ��Ƽ������� ���������� ��ȣ�� �����ϰ� ������ ������� 
//�������ִ� �Լ�
void printstate(uint8_t* data, int dataLen) {
	int i;
	for (i = 0; i < dataLen; i++) {
		if (i != 0 && i % 16 == 0) {
			printf("\n");
		}
		printf("%02x ", data[i]);
	}
}

int main() {
	//��ȣȭ�� data & datalen
	char* MSG = "Hello every body rbrbrb hahaha hahaha hanghanghang hihihi nice to meet you hahaha i am fine thank you ha window linux gpu cpu mac os network database";
	uint32_t msg_size = (uint32_t)strlen(MSG);//'\0'���� ������ ���� return

	//MasterKey in CPU
	uint8_t CPU_MasterKey[16] = { 0x00,0x11,0x22,0x33,0x44,0x55,0x66,0x77,0x88,0x99,0xaa,0xbb,0xcc,0xdd,0xee,0xff };
	//Counter IV
	uint8_t IV[16] = { 0x00, };

	//block -> �޼��� ����� �°� ��ȣȭ�� ��� �� ���� ��
	//blocksize -> ��ȣȭ �ؾ��� ��ϼ��� �°� cuda block ������ ������
	//threadsize 32�� ����(warp���� �۵� ����ؼ� ����)
	int block = ((msg_size - 1) >> 4) + 1;//ceiling ������� ������
	int blocksize = ((block - 1) >> 5) + 1;//ceiling 
	int threadsize = 32;

	//control divergence ���ֱ� ���ؼ� 16*32�� ����� �޼���ũ�� ������
	//new_MSG���� dummy data�� �������
	//���� data ���� ������ dummy data�� �����ϰ� ����
	//���� ���� ����!
	uint32_t new_size = blocksize << 9;//blocksize * 16 * 32
	char* new_MSG = (char*)calloc(new_size, sizeof(uint8_t));
	assert(new_MSG != NULL);
	memcpy(new_MSG, MSG, msg_size * sizeof(uint8_t));

	//���� �޼����� memory coalescing����� ������ �� �ִ� �޸� �迭�� �ٲ��ֱ� ����
	//���ο� �޸� �Ҵ�
	char* COL_MSG = (char*)calloc(new_size, sizeof(uint8_t));
	assert(COL_MSG != NULL);

	int i;

	//���� �޼��� -> memory coalescing����� ������ �� �ִ� �޸� �迭
	//block(��ȣȭ �� block) 0�� index ~ (n-1)�� index ���� ���ʴ�� ��迭
	for (i = 0; i < block; i++) {
		*(COL_MSG + (0 * block + i)) = *(new_MSG + (16 * i + 0));
		*(COL_MSG + (1 * block + i)) = *(new_MSG + (16 * i + 1));
		*(COL_MSG + (2 * block + i)) = *(new_MSG + (16 * i + 2));
		*(COL_MSG + (3 * block + i)) = *(new_MSG + (16 * i + 3));
		*(COL_MSG + (4 * block + i)) = *(new_MSG + (16 * i + 4));
		*(COL_MSG + (5 * block + i)) = *(new_MSG + (16 * i + 5));
		*(COL_MSG + (6 * block + i)) = *(new_MSG + (16 * i + 6));
		*(COL_MSG + (7 * block + i)) = *(new_MSG + (16 * i + 7));
		*(COL_MSG + (8 * block + i)) = *(new_MSG + (16 * i + 8));
		*(COL_MSG + (9 * block + i)) = *(new_MSG + (16 * i + 9));
		*(COL_MSG + (10 * block + i)) = *(new_MSG + (16 * i + 10));
		*(COL_MSG + (11 * block + i)) = *(new_MSG + (16 * i + 11));
		*(COL_MSG + (12 * block + i)) = *(new_MSG + (16 * i + 12));
		*(COL_MSG + (13 * block + i)) = *(new_MSG + (16 * i + 13));
		*(COL_MSG + (14 * block + i)) = *(new_MSG + (16 * i + 14));
		*(COL_MSG + (15 * block + i)) = *(new_MSG + (16 * i + 15));
	}

	//��ȣȭ ���� ���� ������� ��ƿ� ����
	//������: ���� �Ҵ��� �ӵ����� ���鿡�� ���� ����
	//�̸� stack�� ����� ũ���� �޸𸮸� �Ҵ� �� �ʿ��� ��ŭ ���� �����
	//�ӵ����� ���鿡���� �� ȿ����!!!
	uint8_t* enc_msg = (uint8_t*)calloc(msg_size, sizeof(uint8_t));
	assert(enc_msg != NULL);

	//GPU���� ����� �޸� ���� �� �Ҵ�
	uint8_t* GPU_MasterKey = NULL;
	uint8_t* GPU_PT = NULL;
	uint8_t* GPU_ROUNDKEY = NULL;
	uint8_t* GPU_CT = NULL;
	uint8_t* GPU_IV = NULL;
	hipMalloc((void**)&GPU_PT, new_size * sizeof(uint8_t));
	hipMalloc((void**)&GPU_ROUNDKEY, 208 * sizeof(uint8_t));
	hipMalloc((void**)&GPU_CT, new_size * sizeof(uint8_t));
	hipMalloc((void**)&GPU_IV, 16 * sizeof(uint8_t));
	hipMalloc((void**)&GPU_MasterKey, 16 * sizeof(uint8_t));

	//CPU������ �ִ� ����Ÿ -> GPU������ �����鿡 �Ҵ�
	hipMemcpy(GPU_PT, COL_MSG, new_size * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(GPU_IV, IV, 16 * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(GPU_MasterKey, CPU_MasterKey, 16 * sizeof(uint8_t), hipMemcpyHostToDevice);

	//GPU�󿡼� ROUNDKEY ����
	Enc_KEY_Expansion<<<1,1>>>(GPU_MasterKey, GPU_ROUNDKEY);

	//GPU���� ��ȣȭ ���� ������ �� �ֵ���
	//KERNEL �Լ� ȣ��!
	GPU_ARIA_CTR<<<blocksize, threadsize>>>(GPU_PT, GPU_CT, GPU_IV, block, msg_size, GPU_ROUNDKEY);

	//GPU���� �����ϰ� ���� ����� -> CPU�� �ű���!
	//������ control divergence�� ���ϱ� ���ؼ� ����� dummy data �κ��� �߶󳻰�
	//���� ��ȿ�� ����Ÿ ũ���� msg_size��ŭ�� �߶� �ùٸ� ���갪�� ����!
	hipMemcpy(enc_msg, GPU_CT, msg_size * sizeof(uint8_t), hipMemcpyDeviceToHost);

	//�����(enc_msg)��� part
	printstate(enc_msg, msg_size);

	//�޸� ���� �κ�
	hipFree(GPU_PT);
	hipFree(GPU_CT);
	hipFree(GPU_ROUNDKEY);
	hipFree(GPU_IV);
	hipFree(GPU_MasterKey);
	
	free(enc_msg);
	free(COL_MSG);
	free(new_MSG);

	return 0;
}